#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

#define BLOCK_SIZE 256
#define MAX_LOAD_FACTOR 0.9
#define RESHAPE_FACTOR 1.5

// Source: https://burtleburtle.net/bob/hash/integer.html
__device__ uint32_t hashIntegers(uint32_t initialValue) {
    initialValue = (initialValue ^ 61) ^ (initialValue >> 16);
    initialValue = initialValue + (initialValue << 3);
    initialValue = initialValue ^ (initialValue >> 4);
    initialValue = initialValue * 0x27d4eb2d;
    initialValue = initialValue ^ (initialValue >> 15);
    return initialValue;
}

/**
 * The function that calculates the number of blocks needed for
 * running the kernel taking into consideration the number of elements
 * that has to be processed
*/
size_t calculateNumberOfBlocks(int elementsToProcess) {
	size_t blocks_no = elementsToProcess / BLOCK_SIZE;

	// In case the number of elements cannot be divided by the
	// block size we have to add one more block
	if (elementsToProcess % BLOCK_SIZE != 0) {
		blocks_no++;
	}

	return blocks_no;
}

/**
 * Function constructor GpuHashTable
 * Performs init operation for the fields of the GpuHashTable
 * class such as the initial size, the initial load factor,
 * the initial number of elements
 * 
 * It also allocates enough memory on VRAM for the initial
 * size of the hashmap
 */
GpuHashTable::GpuHashTable(int size) {
	this->currentNumberOfElements = 0;
	this->maximumSize = size;
	this->loadFactor = 0.0;
	this->elements = 0;
	hipError_t returnValue;

	glbGpuAllocator->_cudaMalloc((void **) &(this->elements), size * sizeof(GpuHashTableInfo));
	DIE(this->elements == 0, "Error at allocating hashmap in VRAM");

	/* Here we have to initialize the memory to 0 as we will only have to store uint32_t values
	bigger than 0 so when a position has 0 in the hashmap we know that it is not occupied */
	returnValue = hipMemset(this->elements, 0, size * sizeof(GpuHashTableInfo));
	DIE(returnValue != 0, "Error at cuda memset initial values in VRAM");
}

/**
 * Function desctructor GpuHashTable where we free the only allocated
 * memory inside the VRAM which is the hashmap itself
 */
GpuHashTable::~GpuHashTable() {
	hipError_t returnValue = glbGpuAllocator->_cudaFree(this->elements);
	DIE(returnValue != 0, "Error at cuda Free hashmap in VRAM");
}

/**
 * The kernel function used for redimensioning the hashmap to a new size
*/
__global__ void hashmap_reshape(uint32_t oldSize, GpuHashTableInfo *newElements,
								GpuHashTableInfo *oldElements, uint32_t newSize) {
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < oldSize) {
		// If the initial element was 0, we don't have to move it to the new hashmap
		// because all the elements in the new hashmap are already 0
		if (oldElements[i].getKey() == 0) {
			return;
		}

		// Otherwise we just do the hash function for the old element and now we
		// limit it to the new dimension of the hashmap (which is a different
		// value so the final hash is going to be different)
		uint32_t hash = hashIntegers(oldElements[i].getKey());
		uint32_t currentHash = hash % newSize;

		// We check to see if the position where we want to put the key is empty
		// and if it is, we put it there and also put the corresponding value
		int oldKey = atomicCAS(newElements[currentHash].getKeyAddr(), 0, oldElements[i].getKey());

		if (oldKey == 0) {
			atomicExch(newElements[currentHash].getValueAddr(), oldElements[i].getValue());
		} else {
			int nextStep = 1;

			// Otherwise we start doing linear probing until we find an empty position
			// There we put the new key-value pair
			while (nextStep != newSize) {
				unsigned int currentPosition = (hash + nextStep) % newSize;

				oldKey = atomicCAS(newElements[currentPosition].getKeyAddr(), 0, oldElements[i].getKey());

				if (oldKey == 0) {
					atomicExch(newElements[currentPosition].getValueAddr(), oldElements[i].getValue());
					return;
				}

				nextStep++;
			}
		}
	}
}
/**
 * Function reshape
 * Performs resize of the hashtable based on load factor to a new size
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	// Firstly we completly allocate a new hashmap with the new size
	GpuHashTableInfo *newElements = 0;
	hipError_t returnValue = glbGpuAllocator->_cudaMalloc((void **) &newElements,
														   numBucketsReshape * sizeof(GpuHashTableInfo));
	DIE(returnValue != 0, "Error at hipMalloc for the new elements in reshape");

	// Then we initialize it to 0 everywhere because we can only have positive values
	// so 0 values means that the position has not been occupied yet
	returnValue = hipMemset(newElements, 0, numBucketsReshape * sizeof(GpuHashTableInfo));
	DIE(returnValue != 0, "Error at cuda memset initial values in VRAM");

	size_t blocks_no = calculateNumberOfBlocks(this->maximumSize);
	// Then we call the kernel with blocks number being maximum size of the hashmap over
	// the size of one block because even though we can have less elements in the hashmap
	// than the maximum size we have no ideea at what position those are located so we 
	// actually have to traverse the whole hashmap
	hashmap_reshape<<<blocks_no, BLOCK_SIZE>>>(this->maximumSize, newElements,
										       this->elements, numBucketsReshape);
	returnValue = hipDeviceSynchronize();
	DIE(returnValue != 0, "Error at cuda device synchronize after reshape kernel");

	// Here we free the old hashmap and update the fields with their new values
	returnValue = glbGpuAllocator->_cudaFree(this->elements);
	DIE(returnValue != 0, "Error at cuda Free for the old elements in VRAM");

	this->elements = newElements;
	this->maximumSize = numBucketsReshape;
	this->loadFactor = (double) this->currentNumberOfElements / this->maximumSize;
}

/**
 * The kernel function used to insert a vector of keys and values
 * into the hashmap using the GPU
*/
__global__ void hashmap_insert(const int *keys, const int *values,
							   const int maximumSize, GpuHashTableInfo *elements,
							   unsigned int *newElements, const int numKeys) {
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < numKeys) {
		// Firstly we calculate the current hash of the key and limit it to the
		// current maximum size of the hashmap and try to put it at the specific position
		uint32_t hash = hashIntegers(keys[i]);
		uint32_t currentHash = hash % maximumSize;

		int oldKey = atomicCAS(elements[currentHash].getKeyAddr(), 0, keys[i]);
		int oldValue;

		// If the position was previously empty or it contained exactly the same key
		// that we are trying to put right now, it means that we have to also update
		// the value of that specific key
		if (oldKey == 0 || oldKey == keys[i]) {
			oldValue = atomicExch(elements[currentHash].getValueAddr(), values[i]);

			// If the old value was 0 it means that it is a completly new key-value
			// pair added to the hashmap so we also have to increment the number of
			// new values added to the hashmap
			if (oldValue == 0) {
				atomicInc(newElements, maximumSize);
			}

			return;
		} else {
			int nextStep = 1;

			// Otherwise we start doing linear probing until we find an empty position
			// There we put the new key-value pair
			while (nextStep != maximumSize) {
				unsigned int currentPosition = (hash + nextStep) % maximumSize;

				oldKey = atomicCAS(elements[currentPosition].getKeyAddr(), 0, keys[i]);

				if (oldKey == 0 || oldKey == keys[i]) {
					oldValue = atomicExch(elements[currentPosition].getValueAddr(), values[i]);

					// If the old value was 0 it means that it is a completly new key-value
					// pair added to the hashmap so we also have to increment the number of
					// new values added to the hashmap
					if (oldValue == 0) {
						atomicInc(newElements, maximumSize);
					}

					return;
				}

				nextStep++;
			}
		}
	}

	return;
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	// Here, firstly we verify what is going to be the maximum load factor if we are going
	// to add all the given keys to the hashmap. I am saying here the `maximum` load factor
	// because some of the keys might be identical to some of the keys that are already
	// inside the hashmap so in that case, we just have to update the value so no more keys
	// added to the hashmap. If the maximum possible load factor is going to exceed for example
	// 90%, we will have to resize it to 1.5 * currentSize in order to have a balanced hashmap
	// and not to fall with the load factor under 50%
	double possibleMaxLoadFactor = (double) (this->currentNumberOfElements + numKeys) / this->maximumSize;
	if (possibleMaxLoadFactor >= MAX_LOAD_FACTOR) {
		this->reshape(this->maximumSize * RESHAPE_FACTOR);
	}

	int *device_keys = 0;
	int *device_values = 0;

	// Here we copy the keys and the values from host to device
	glbGpuAllocator->_cudaMalloc((void **) &device_keys, numKeys * sizeof(int));
	glbGpuAllocator->_cudaMalloc((void **) &device_values, numKeys * sizeof(int));
	DIE(device_keys == 0 || device_values == 0, "Error at hipMalloc for keys/values in insert");

	hipError_t returnValue = hipMemcpy(device_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	DIE(returnValue != 0, "Error at cuda memcpy host keys to device keys in VRAM");

	returnValue = hipMemcpy(device_values, values, numKeys * sizeof(int), hipMemcpyHostToDevice);
	DIE(returnValue != 0, "Error at cuda memcpy host values to device values in VRAM");

	// We calculate the blocks number as being the number of keys over the size of one block
	// that is because we have to add maximum numKeys keys inside the hashmap
	size_t blocks_no = calculateNumberOfBlocks(numKeys);

	// This is going to be a hipMallocManaged integer value that is going to be allocated in the
	// unified memory. This is happening because it is going to be changed by the insert kernel
	// each time when a new key-pair value is added in the hashmap and it is also going to be
	// verified by the host in order to know how many actual key were added durring the kernel exec
	unsigned int *newElements = 0;
	glbGpuAllocator->_cudaMallocManaged((void **) &newElements, sizeof(unsigned int));
	DIE(newElements == 0, "Error at hipMalloc for the number of new inserted elements");
	*newElements = 0;

	hashmap_insert<<<blocks_no, BLOCK_SIZE>>>(device_keys, device_values, this->maximumSize,
	 										  this->elements, newElements, numKeys);
	returnValue = hipDeviceSynchronize();
	DIE(returnValue != 0, "Error at cuda device synchronize after insert kernel");

	// Here, if there were any new key-value pairs added to the hashmap we increment the
	// current number of elements and recalculate the load factor
	if (*newElements != 0) {
		this->currentNumberOfElements += *newElements;
		this->loadFactor = (double) this->currentNumberOfElements / this->maximumSize;
	}

	// We free the device keys, values and the new elements int value and exit the function
	returnValue = glbGpuAllocator->_cudaFree(device_keys);
	DIE(returnValue != 0, "Error at cuda Free device keys in VRAM");

	returnValue = glbGpuAllocator->_cudaFree(device_values);
	DIE(returnValue != 0, "Error at cuda Free device values in VRAM");

	returnValue = glbGpuAllocator->_cudaFree(newElements);
	DIE(returnValue != 0, "Error at cuda Free unified memory for the new inserted elements");

	return true;
}

/**
 * The kernel function used to get a vector of values starting from a vector
 * of keys from the hashmap
*/
__global__ void hashmap_get(const int *keys, int *result_values,
					        const int maximumSize, GpuHashTableInfo *elements,
					        const int numKeys)
{
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	// Here we do the same operation as in inserting. We firstly just calculate
	// the hash of the current key that we are trying to find. And then we try
	// to see if the key from the position of the hash is the one we are actually
	// searching for. If it is, we just take the value from the same position and return.
	if (i < numKeys) {
		uint32_t hash = hashIntegers(keys[i]);
		uint32_t currentHash = hash % maximumSize;

		int oldKey = elements[currentHash].getKey();
		int oldValue;

		if (oldKey == keys[i]) {
			oldValue = elements[currentHash].getValue();
			result_values[i] = oldValue;
			return;

		} else {
			int nextStep = 1;

			// Otherwise we start doing linear probing until we find a position
			// where the key from that position is the same with the key we are
			// searching for
			while (nextStep != maximumSize) {
				unsigned int currentPosition = (hash + nextStep) % maximumSize;

				oldKey = elements[currentPosition].getKey();

				if (oldKey == keys[i]) {
					oldValue = elements[currentPosition].getValue();
					result_values[i] = oldValue;
					return;
				}

				nextStep++;
			}
		}
	}
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	int *device_keys = 0;
	int *result_values = 0;

	// Firstly we allocate the device_keys vector and copy the keys from the host to device.
	hipError_t returnValue = glbGpuAllocator->_cudaMalloc((void **) &device_keys, numKeys * sizeof(int));
	DIE(returnValue != 0, "Error at cuda malloc for the device keys in get batch");

	returnValue = hipMemcpy(device_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	DIE(returnValue != 0, "Error at cuda memcpy for the keys into device keys in get batch");

	// Then, here we allocate a result_values vector with hipMallocManaged because it is going to
	// be populated by the kernel but it is going to be returned from this host function and also
	// used further in main.
	returnValue = glbGpuAllocator->_cudaMallocManaged((void **) &result_values, numKeys * sizeof(int));
	DIE(returnValue != 0, "Error at cuda malloc for the result values in get batch");

	// We calculate the blocks number as being the number of keys over the size of one block
	// that is because we have to get exactly numKeys keys from the hashmap
	size_t blocks_no = calculateNumberOfBlocks(numKeys);

	// We execute the kernel, free the device keys and return the newly populated result_values
	// from the kernel.
	hashmap_get<<<blocks_no, BLOCK_SIZE>>>(device_keys, result_values, this->maximumSize,
										   this->elements, numKeys);
	returnValue = hipDeviceSynchronize();
	DIE(returnValue != 0, "Error at cuda device synchronize after insert kernel");

	returnValue = glbGpuAllocator->_cudaFree(device_keys);
	DIE(returnValue != 0, "Error at cuda Free device values in VRAM");

	return result_values;
}
